#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<hip/hip_runtime.h>
//#include "../include_cpp/structures.h"


__device__
inline float euclidian_norm2D(float x, float y)
{
    return sqrt(pow(x,2) + pow(y,2));
}

__device__
inline bool is_neighbour2D(float c_x, float c_y, float p_x, float p_y, float bandwidth)
{
    //DEBUG
    //std::cout << "distance: " << euclidian_norm2D(c_x - p_x, c_y - p_y)<<std::endl;
    //
    return euclidian_norm2D(c_x - p_x, c_y - p_y) < bandwidth? true:false;
}

__global__
void centroid_convergence_kernel(float* d_centroid_x, float* d_centroid_y, float* d_points_x, float* d_points_y, int num_points)
{
    int thread_id = blockDim.x*blockIdx.x + threadIdx.x;
    bool check = false;//automatic variable
    //float bandwidth = BANDWIDTH;
    float meanshift_x = 0;
    float meanshift_y = 0;
    int num_neighbours = 0;
    if(thread_id < num_points)
    {    
        for(int i = 0; i < num_points; i++)
        {
            check = is_neighbour2D(d_centroid_x[thread_id], d_centroid_y[thread_id], d_points_x[i], d_points_y[i], BANDWIDTH);
            if(check)
            {
                meanshift_x += d_points_x[i];
                meanshift_y += d_points_y[i];
                num_neighbours += 1;
            }
        }

        d_centroid_x[thread_id] = meanshift_x / num_neighbours;
        d_centroid_y[thread_id] = meanshift_y / num_neighbours;
    }
}




void meanshift_convergence(dataset2D centroids, dataset2D points, int num_points)
{
    float *d_centroid_x;
    float *d_centroid_y;

    float *d_points_x;
    float *d_points_y;

    float* d_meanshift_x;
    float* d_meanshift_y;

    int* d_num_neighbours;

    hipMalloc((void**) &d_centroid_x, sizeof(float)*num_points);
    hipMemcpy(d_centroid_x, centroids.x, sizeof(float)*num_points, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_centroid_y, sizeof(float)*num_points);
    hipMemcpy(d_centroid_y, centroids.y, sizeof(float)*num_points, hipMemcpyHostToDevice);

    hipMalloc((void**) &d_points_x, sizeof(float)*num_points);
    hipMemcpy(d_points_x, points.x, sizeof(float)*num_points, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_points_y, sizeof(float)*num_points);
    hipMemcpy(d_points_y, points.y, sizeof(float)*num_points, hipMemcpyHostToDevice);

    //KERNEL CALL
    //We will have a 1d block
    for(int i = 0; i < MAX_ITERS; i++)
    {
        centroid_convergence_kernel<<<ceil(num_points/(float)BLOCK_DIM), BLOCK_DIM>>>(d_centroid_x, d_centroid_y, d_points_x, d_points_y, num_points);
    }
    

    hipMemcpy(centroids.x, d_centroid_x, sizeof(float)*num_points, hipMemcpyDeviceToHost);
    hipMemcpy(centroids.y, d_centroid_y, sizeof(float)*num_points, hipMemcpyDeviceToHost);

    hipFree(d_centroid_x);
    hipFree(d_centroid_y);
    hipFree(d_points_x);
    hipFree(d_points_y);

    return;
}

__global__
void merge_clusters_kernel(float* d_centroid_xi, float* d_centroid_yi, float* d_centroid_x, float* d_centroid_y, int* d_points_labels, int num_points, int* label)
{
    int thread_id = blockDim.x*blockIdx.x + threadIdx.x;
    bool check = false;
    if (thread_id < num_points)
    {
        if(d_points_labels[thread_id] == -1)
            check = is_neighbour2D(*d_centroid_xi, *d_centroid_xi, d_centroid_x[thread_id], d_centroid_y[thread_id], BANDWIDTH);
            if (check)
                d_points_labels[thread_id] = *label;
    }
}

void meanshift_merge(dataset2D centroids, int num_points)
{
    float* d_centroid_x;
    float* d_centroid_y;
    int* d_centroids_labels;

    int* label;
    int* d_label;

    hipMalloc((void**) &d_centroid_x, sizeof(float)*num_points);
    hipMemcpy(d_centroid_x, centroids.x, sizeof(float)*num_points, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_centroid_y, sizeof(float)*num_points);
    hipMemcpy(d_centroid_y, centroids.y, sizeof(float)*num_points, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_centroids_labels, sizeof(int)*num_points);
    hipMemcpy(d_centroids_labels, centroids.labels, sizeof(int)*num_points, hipMemcpyHostToDevice);

    hipMalloc((void**) &d_label, sizeof(int));
    hipMemset(d_label, 0, sizeof(int));

    //KERNEL CALL
    //We will have a 1d block
    for(int i = 0; i < num_points; i++)
    {
        merge_clusters_kernel<<<ceil(num_points/(float)BLOCK_DIM), BLOCK_DIM>>>(&d_centroid_x[i], &d_centroid_y[i], d_centroid_x, d_centroid_y, d_centroids_labels, num_points, d_label);
        hipMemcpy(label, d_label, sizeof(int), hipMemcpyDeviceToHost);
        label += 1;
        hipMemcpy(d_label, label, sizeof(int), hipMemcpyHostToDevice);
        //*d_label = *d_label + 1;
    }

    hipMemcpy(centroids.labels, d_centroids_labels, sizeof(float)*num_points, hipMemcpyDeviceToHost);

    hipFree(d_centroid_x);
    hipFree(d_centroid_y);
    hipFree(d_centroids_labels);
    hipFree(d_label);

    return;
}


//---------------------------------------------------------------------------------------------------


__global__
void centroid_convergence_kernel_2(float* d_centroid_x, float* d_centroid_y, float* d_points_x, float* d_points_y, int num_points, float* d_ms_x, float* d_ms_y, int* d_num_neighbours)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    bool check = false;//automatic variable
    //float bandwidth = BANDWIDTH;
    if(i < num_points)
    {
        check = is_neighbour2D(*d_centroid_x, *d_centroid_y, d_points_x[i], d_points_y[i], BANDWIDTH);
        if(check)
        {
            //accumulate the coordinates of the point
            //atomic addition to accumulate the coordinates
            atomicAdd(d_ms_x, d_points_x[i]);
            atomicAdd(d_ms_y, d_points_y[i]);
            atomicAdd(d_num_neighbours, 1);
            //check = false;
        } 
    }  
}

void meanshift_convergence_2(dataset2D centroids, dataset2D points, int num_points)
{
    float *d_centroid_x;
    float *d_centroid_y;

    float *d_points_x;
    float *d_points_y;

    float* meanshift_x;
    float* meanshift_y;

    float* d_meanshift_x;
    float* d_meanshift_y;

    int* num_neighbours;
    int* d_num_neighbours;

    hipMalloc((void**) &d_centroid_x, sizeof(float)*num_points);
    hipMemcpy(d_centroid_x, centroids.x, sizeof(float)*num_points, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_centroid_y, sizeof(float)*num_points);
    hipMemcpy(d_centroid_y, centroids.y, sizeof(float)*num_points, hipMemcpyHostToDevice);

    hipMalloc((void**) &d_points_x, sizeof(float)*num_points);
    hipMemcpy(d_points_x, points.x, sizeof(float)*num_points, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_points_y, sizeof(float)*num_points);
    hipMemcpy(d_points_y, points.y, sizeof(float)*num_points, hipMemcpyHostToDevice);

    //Meanshift x and y
    hipMalloc((void**) &d_meanshift_x, sizeof(float));
    hipMemset(d_meanshift_x, 0, sizeof(float));
    hipMalloc((void**) &d_meanshift_y, sizeof(float));
    hipMemset(d_meanshift_y, 0, sizeof(float));

    //Nukmber of Neighbours
    hipMalloc((void**) &d_num_neighbours, sizeof(int));
    hipMemset(d_num_neighbours, 0, sizeof(int));

    //KERNEL CALL
    //We will have a 1d block
    bool converged = false;
    for(int i = 0; i < num_points; i++)
    {
        converged = false;
        hipMemset(d_meanshift_x, 0, sizeof(float));
        hipMemset(d_meanshift_y, 0, sizeof(float));

        for(int j = 0; j < MAX_ITERS; j++)
        {
            centroid_convergence_kernel_2<<<ceil(num_points/(float)BLOCK_DIM), BLOCK_DIM>>>(&d_centroid_x[i], &d_centroid_y[i], d_points_x, d_points_y, num_points, d_meanshift_x, d_meanshift_y, d_num_neighbours);
            
            hipMemcpy(meanshift_x, d_meanshift_x, sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(meanshift_y, d_meanshift_y, sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(num_neighbours, d_num_neighbours, sizeof(int), hipMemcpyDeviceToHost);

            *meanshift_x = *meanshift_x/(float)*num_neighbours;
            *meanshift_y = *meanshift_y/(float)*num_neighbours;

            /*centroids.x[i] = *meanshift_x;
            centroids.y[i] = *meanshift_y;
            hipMemcpy(&d_centroid_x[i], &centroids.x[i], sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(&d_centroid_y[i], &centroids.y[i], sizeof(float), hipMemcpyHostToDevice);*/

            hipMemcpy(&d_centroid_x[i], meanshift_x, sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(&d_centroid_y[i], meanshift_y, sizeof(float), hipMemcpyHostToDevice);

            hipMemset(d_meanshift_x, 0, sizeof(float));
            hipMemset(d_meanshift_y, 0, sizeof(float));
            hipMemset(d_num_neighbours, 0, sizeof(int));
        }  
    }

    hipMemcpy(centroids.x, d_centroid_x, sizeof(float)*num_points, hipMemcpyDeviceToHost);
    hipMemcpy(centroids.y, d_centroid_y, sizeof(float)*num_points, hipMemcpyDeviceToHost);

    hipFree(d_centroid_x);
    hipFree(d_centroid_y);
    hipFree(d_points_x);
    hipFree(d_points_y);
    hipFree(d_meanshift_x);
    hipFree(d_meanshift_y);
    hipFree(d_num_neighbours);

    return;
}